#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>

#include "hip/hip_runtime.h"
#include "../hip/hip_runtime_api.h"
#include "opencv2/cudev.hpp"
using namespace cv::cuda;


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Utility Functions
struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float ElapsedMillis()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////


// CUDA kernel for outputing the final ARGB output from NV12;

__global__ void NV12ToARGB(uint32_t *srcImage,   size_t nv12Pitch,
                             uint32_t *dstImage,   size_t argbPitch,
                             uint32_t width,       uint32_t height)
{
    int32_t x, y;
    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = nv12Pitch;
    uint32_t dstImagePitch   = argbPitch >> 2;
    uint8_t *srcImageU8     = (uint8_t *)srcImage;


    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width)
        return; //x = width - 1;

    if (y >= height)
        return; // y = height - 1;

    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values
	// Also convert to 10-bit value (by shifting left by 2)
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32_t chromaOffset    = processingPitch * height;
    int32_t y_chroma = y >> 1;

    if (y & 1)  // odd scanline ?
    {
        uint32_t chromaCb;
        uint32_t chromaCr;

        chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
        chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << 12);
        yuv101010Pel[0] |= (chromaCr << 22);

        yuv101010Pel[1] |= (chromaCb << 12);
        yuv101010Pel[1] |= (chromaCr << 22);
    }
    else
    {
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << 12);
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << 22);

        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << 12);
        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << 22);
    }

    // this steps performs the color conversion
    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] &   0x3FF);
    yuvi[1] = ((yuv101010Pel[0] >>  10)       & 0x3FF);
    yuvi[2] = ((yuv101010Pel[0] >> 20) & 0x3FF);

    yuvi[3] = (yuv101010Pel[1] &   0x3FF);
    yuvi[4] = ((yuv101010Pel[1] >>  10)       & 0x3FF);
    yuvi[5] = ((yuv101010Pel[1] >> 20) & 0x3FF);

    // YUV to RGB Transformation conversion
   	float luma[2], chromaCb[2], chromaCr[2];	
	uint32_t ARGB[2];

	// Prepare for hue adjustment
    luma[0]     = (float)yuvi[0];
    chromaCb[0] = (float)((int32_t)yuvi[1] - 512.0f);
    chromaCr[0] = (float)((int32_t)yuvi[2] - 512.0f);

	luma[1]     = (float)yuvi[3];
    chromaCb[1] = (float)((int32_t)yuvi[4] - 512.0f);
    chromaCr[1] = (float)((int32_t)yuvi[5] - 512.0f);

    // Convert YUV To RGB with hue adjustment
    red[0]  = (luma[0] * 1.1644f) +						       (chromaCr[0] * 1.5960f);           
    green[0]= (luma[0] * 1.1644f) + (chromaCb[0] * -0.3918f) + (chromaCr[0] * -0.8130f);            
    blue[0] = (luma[0] * 1.1644f) + (chromaCb[0] * 2.0172f);

	red[1]  = (luma[1] * 1.1644f) +						       (chromaCr[1] * 1.5960f);           
    green[1]= (luma[1] * 1.1644f) + (chromaCb[1] * -0.3918f) + (chromaCr[1] * -0.8130f);            
    blue[1] = (luma[1] * 1.1644f) + (chromaCb[1] * 2.0172f);


    // Clamp the results to RGBA to 10 bits
	if(red[0]<0.0f)   red[0]   = 0.0f;	if(red[0]>1023.0f)   red[0]   = 1023.0f;
	if(green[0]<0.0f) green[0] = 0.0f;	if(green[0]>1023.0f) green[0] = 1023.0f;
	if(blue[0]<0.0f)  blue[0]  = 0.0f;	if(blue[0]>1023.0f)  blue[0]  = 1023.0f;

	if(red[1]<0.0f)   red[1]   = 0.0f;	if(red[1]>1023.0f)   red[1]   = 1023.0f;
	if(green[1]<0.0f) green[1] = 0.0f;	if(green[1]>1023.0f) green[1] = 1023.0f;
	if(blue[1]<0.0f)  blue[1]  = 0.0f;	if(blue[1]>1023.0f)  blue[1]  = 1023.0f;

	
	// Convert to 8 bit unsigned integers per color component with alpha = 0xff000000
    ARGB[0] = (((uint32_t)blue[0]  >> 2) | (((uint32_t)green[0] >> 2) << 8) | (((uint32_t)red[0]   >> 2) << 16) | 0xff000000);
	ARGB[1] = (((uint32_t)blue[1]  >> 2) | (((uint32_t)green[1] >> 2) << 8) | (((uint32_t)red[1]   >> 2) << 16) | 0xff000000);
	
	// copy to destination image
    dstImage[y * dstImagePitch + x     ] = ARGB[0];
    dstImage[y * dstImagePitch + x + 1 ] = ARGB[1];

    __syncthreads();  // wait for all threads to complete
}

extern "C"
void cuda_NV12ToARGB(hipDeviceptr_t nv12ImagePtr, hipDeviceptr_t argbImagePtr,  
					   uint32_t width,  uint32_t height, uint32_t nv12Pitch, uint32_t argbPitch, hipStream_t stream)
{
	uint32_t * srcImage = (uint32_t*)nv12ImagePtr;
    uint32_t * dstImage = (uint32_t*)argbImagePtr;
       
	dim3 threadsPerBlock(32,16);  // 32x16 = 512 threads per block

	// NOTE the 2 here ------V   and here ------------V which are there because each thread processes 2 pixels at a time 
	dim3 numBlocks((width + (2*threadsPerBlock.x)-1)/(2*threadsPerBlock.x), (height + threadsPerBlock.y-1)/threadsPerBlock.y); 
	NV12ToARGB<<<numBlocks,threadsPerBlock,0,stream>>>(srcImage,nv12Pitch,dstImage,argbPitch,width,height);
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////

// CUDA kernel for outputing the final RGB output from NV12;

__global__ void NV12ToRGB(uint32_t *srcImage,   size_t nv12Pitch,
                             uint8_t *dstImage,   size_t rgbPitch,
                             uint32_t width,       uint32_t height)
{
    int32_t x, y;
    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = nv12Pitch;
    uint32_t dstImagePitch   = rgbPitch;
    uint8_t *srcImageU8     = (uint8_t *)srcImage;


    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width) return; 
    if (y >= height) return; 

    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values
	// Also convert to 10-bit value (by shifting left by 2)
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32_t chromaOffset    = processingPitch * height;
    int32_t y_chroma = y >> 1;

    if (y & 1)  // odd scanline ?
    {
        uint32_t chromaCb;
        uint32_t chromaCr;

        chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
        chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << 12);
        yuv101010Pel[0] |= (chromaCr << 22);

        yuv101010Pel[1] |= (chromaCb << 12);
        yuv101010Pel[1] |= (chromaCr << 22);
    }
    else
    {
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << 12);
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << 22);

        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << 12);
        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << 22);
    }

    // this steps performs the color conversion
    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] &   0x3FF);
    yuvi[1] = ((yuv101010Pel[0] >>  10)       & 0x3FF);
    yuvi[2] = ((yuv101010Pel[0] >> 20) & 0x3FF);

    yuvi[3] = (yuv101010Pel[1] &   0x3FF);
    yuvi[4] = ((yuv101010Pel[1] >>  10)       & 0x3FF);
    yuvi[5] = ((yuv101010Pel[1] >> 20) & 0x3FF);

    // YUV to RGB Transformation conversion
   	float luma[2], chromaCb[2], chromaCr[2];

	// Prepare for hue adjustment
    luma[0]     = (float)yuvi[0];
    chromaCb[0] = (float)((int32_t)yuvi[1] - 512.0f);
    chromaCr[0] = (float)((int32_t)yuvi[2] - 512.0f);

	luma[1]     = (float)yuvi[3];
    chromaCb[1] = (float)((int32_t)yuvi[4] - 512.0f);
    chromaCr[1] = (float)((int32_t)yuvi[5] - 512.0f);

    // Convert YUV To RGB with hue adjustment
    red[0]  = (luma[0] * 1.1644f) +						       (chromaCr[0] * 1.5960f);           
    green[0]= (luma[0] * 1.1644f) + (chromaCb[0] * -0.3918f) + (chromaCr[0] * -0.8130f);            
    blue[0] = (luma[0] * 1.1644f) + (chromaCb[0] * 2.0172f);

	red[1]  = (luma[1] * 1.1644f) +						       (chromaCr[1] * 1.5960f);           
    green[1]= (luma[1] * 1.1644f) + (chromaCb[1] * -0.3918f) + (chromaCr[1] * -0.8130f);            
    blue[1] = (luma[1] * 1.1644f) + (chromaCb[1] * 2.0172f);


    // Clamp the results to RGB to 10 bits
	if(red[0]<0.0f)   red[0]   = 0.0f;	if(red[0]>1023.0f)   red[0]   = 1023.0f;
	if(green[0]<0.0f) green[0] = 0.0f;	if(green[0]>1023.0f) green[0] = 1023.0f;
	if(blue[0]<0.0f)  blue[0]  = 0.0f;	if(blue[0]>1023.0f)  blue[0]  = 1023.0f;

	if(red[1]<0.0f)   red[1]   = 0.0f;	if(red[1]>1023.0f)   red[1]   = 1023.0f;
	if(green[1]<0.0f) green[1] = 0.0f;	if(green[1]>1023.0f) green[1] = 1023.0f;
	if(blue[1]<0.0f)  blue[1]  = 0.0f;	if(blue[1]>1023.0f)  blue[1]  = 1023.0f;
		
	// copy to destination image
	uint32_t offset = (y * dstImagePitch) + (x * 3);
	
	//uint32_t offset = (y * dstImagePitch) + (blockIdx.x * blockDim.x + threadIdx.x) * 6;

	dstImage[offset + 0] = (uint8_t)((uint32_t)red[0] >> 2);
	dstImage[offset + 1] = (uint8_t)((uint32_t)green[0] >> 2);
	dstImage[offset + 2] = (uint8_t)((uint32_t)blue[0] >> 2);

	dstImage[offset + 3] = (uint8_t)((uint32_t)red[1] >> 2);
	dstImage[offset + 4] = (uint8_t)((uint32_t)green[1] >> 2);
	dstImage[offset + 5] = (uint8_t)((uint32_t)blue[1] >> 2);

    __syncthreads();  // wait for all threads to complete
}

extern "C"
void cuda_NV12ToRGB(hipDeviceptr_t nv12ImagePtr, hipDeviceptr_t rgbImagePtr,  
					  uint32_t width,  uint32_t height, uint32_t nv12Pitch, uint32_t rgbPitch)
{
	uint32_t * srcImage = (uint32_t*)nv12ImagePtr;
    uint8_t * dstImage = (uint8_t*)rgbImagePtr;
       
	dim3 threadsPerBlock(32,16);  // 32x16 = 512 threads per block

	// NOTE the 2 here ------V   and here ------------V which are there because each thread processes 2 pixels at a time 
	dim3 numBlocks((width + (2*threadsPerBlock.x)-1)/(2*threadsPerBlock.x), (height + threadsPerBlock.y-1)/threadsPerBlock.y); 
	NV12ToRGB<<<numBlocks,threadsPerBlock>>>(srcImage,nv12Pitch,dstImage,rgbPitch,width,height);
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void CudaToGpuMat(uint32_t *destImage, uint32_t *sourceImage, uint32_t width, uint32_t height)
{
    // destImage should be a pointer to the GpuMat data (i.e. GpuMat.ptr())
    // sourceImage should be a pointer to an RGBA image in GPU memory (BGRA format)
    // width is the width in pixels
    // height is the height in pixels

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        uint32_t idx = y * width + x;

		// convert from RGBA to BGRA
		uint32_t value = sourceImage[idx];		
		
		uint8_t *ptr = (uint8_t*)&value;

		uint8_t red = ptr[0];
		uint8_t green = ptr[1];
		uint8_t blue = ptr[2];
		uint8_t alpha = ptr[3];

		ptr[0] = blue;
		ptr[1] = green;
		ptr[2] = red;
		ptr[3] = alpha;
		
        destImage[idx] = value;
    }
}

extern "C"
void cuda_CudaToGpuMat(hipDeviceptr_t GpuMatDataPtr, hipDeviceptr_t CudaImagePtr,  uint32_t width,  uint32_t height)
{
	uint32_t * srcImage = (uint32_t*)CudaImagePtr;
    uint32_t * dstImage = (uint32_t*)GpuMatDataPtr;
       
	dim3 threadsPerBlock(32,16);  // 32x16 = 512 threads per block
	dim3 numBlocks((width + threadsPerBlock.x-1)/threadsPerBlock.x, (height + threadsPerBlock.y-1)/threadsPerBlock.y); 
	CudaToGpuMat<<<numBlocks,threadsPerBlock>>>(dstImage,srcImage,width,height);
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void NV12PanelToComposite(uint8_t* PanelPtr, uint8_t* CompositePtr,			// gpu memory pointers to panel image and composite image
										uint32_t PanelWidth, uint32_t PanelHeight,			// pixel width/height of panel
										uint32_t CompositeWidth, uint32_t CompositeHeight,	// pixel width/height of composite
										uint32_t PanelRow, uint32_t PanelColumn)			// target location of panel within composite
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

	// reassign variables just to shorten the name for convenience
	uint32_t r = PanelRow;			// row of panel inside composite
	uint32_t c = PanelColumn;		// column of panel inside composite
	uint32_t WP = PanelWidth;		// pixel width of panel
	uint32_t HP = PanelHeight;		// pixel height of panel
	uint32_t WC = CompositeWidth;	// pixel width of composite
	uint32_t HC = CompositeHeight;	// pixel height of composite

	uint32_t X = c * WP + x;		// column of pixel inside composite
	uint32_t Y = r * HP + y;		// row of pixel inside composite

	uint32_t n = (y * WP) + x;		// index into Luma array of panel
	uint32_t N = (Y * WC) + X;		// index into Luma array of composite

	uint8_t* LP = PanelPtr;			// pointer to start of Luma array of panel
	uint8_t* LC = CompositePtr;     // pointer to start of Luma array of composite
	uint8_t* CP = LP + (WP * HP);	// pointer to start of Chroma array of panel
	uint8_t* CC = LC + (WC * WP);	// pointer to start of Chroma array of composite

	
	LC[N] = LP[n];	// copy Luma value from panel to composite

	if((y%2) == 0)	// only on even rows, copy Chroma from panel to composite (only half as many Chroma rows as Luma rows)
	{
		uint32_t nc = (WP * HP) + (y/2 * WP) + x;	// index into Chroma array of panel
		uint32_t Nc = (WC * HC) + (Y/2 * WC) + X;	// index into Chroma array of composite

		CC[Nc] = CP[nc];	// copy Chroma value from panel to composite
	}

	__syncthreads();	// wait for all threads to complete that are copying the panel to the composite
}

extern "C"
void cuda_NV12PanelToComposite(hipDeviceptr_t PanelCUPtr, hipDeviceptr_t CompositeCUPtr,  
							   uint32_t PanelWidth,  uint32_t PanelHeight,
							   uint32_t CompositeWidth, uint32_t CompositeHeight,
							   uint32_t PanelRow, uint32_t PanelColumn)
{
	uint8_t * PanelPtr = (uint8_t*)PanelCUPtr;
    uint8_t * CompositePtr = (uint8_t*)CompositeCUPtr;
       
	dim3 threadsPerBlock(32,16);  // 32x16 = 512 threads per block
	dim3 numBlocks((PanelWidth + threadsPerBlock.x-1)/threadsPerBlock.x, (PanelHeight + threadsPerBlock.y-1)/threadsPerBlock.y); 
	NV12PanelToComposite<<<numBlocks,threadsPerBlock>>>(PanelPtr,CompositePtr,PanelWidth,PanelHeight,
													    CompositeWidth,CompositeHeight,PanelRow,PanelColumn);
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void CopyCudaArrayToD3D9Texture(uint8_t *dest, uint8_t *source, uint16_t pitch, uint16_t width, uint16_t height)
{
	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

	// make sure we don't try to operate outside the image
	if(x>=width) return;
	if(y>=height) return;

	// calc position of pixel in cuda array (remember that pitch may not equal width)
	//uint32_t nD = ((height-y)*pitch) + (x*4);
	uint32_t nD = (y*pitch) + (x * 4);
	uint32_t nS = (y*width*4) + (x*4);

	// copy data
	dest[nD]   = source[nS];
	dest[nD+1] = source[nS+1];
	dest[nD+2] = source[nS+2];
	dest[nD+3] = source[nS+3];	
}


extern "C"
void cuda_CopyCudaArrayToD3D9Texture(hipDeviceptr_t pDest, hipDeviceptr_t pSource, uint16_t pitch, uint16_t width, uint16_t height, hipStream_t stream)
{

	uint8_t* dest = (uint8_t*)pDest;
	uint8_t* source = (uint8_t*)pSource;

	//dim3 threadsPerBlock(32,16);  // 32x16 = 512 threads per block	
	dim3 threadsPerBlock(32, 32);  // 32x16 = 512 threads per block	
	dim3 numBlocks((width + threadsPerBlock.x-1)/threadsPerBlock.x, (height + threadsPerBlock.y-1)/threadsPerBlock.y); 

	CopyCudaArrayToD3D9Texture<<<numBlocks,threadsPerBlock,0,stream>>>(dest,source,pitch,width,height);		

}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////

